#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <sys/time.h>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

#include "des.h"
#include "utils.h"

using namespace std;


__device__ int ip_cuda[] = {
        58,50,42,34,26,18,10, 2,
        60,52,44,36,28,20,12, 4,
        62,54,46,38,30,22,14, 6,
        64,56,48,40,32,24,16, 8,
        57,49,41,33,25,17, 9, 1,
        59,51,43,35,27,19,11, 3,
        61,53,45,37,29,21,13, 5,
        63,55,47,39,31,23,15, 7,
};

__device__ int fp_cuda[64] = {
        40, 8,48,16,56,24,64,32,
        39, 7,47,15,55,23,63,31,
        38, 6,46,14,54,22,62,30,
        37, 5,45,13,53,21,61,29,
        36, 4,44,12,52,20,60,28,
        35, 3,43,11,51,19,59,27,
        34, 2,42,10,50,18,58,26,
        33, 1,41, 9,49,17,57,25,
};


__device__ int per_cuda[32]={
        16, 7, 20,21,
        29, 12, 28, 17,
        1, 15, 23, 26,
        5, 18, 31, 10,
        2, 8, 24, 14,
        32, 27, 3, 9,
        19, 13, 30, 6,
        22, 11, 4, 25
};

__device__ int s_cuda[8][4][16] = {
        {
                14,4,13,1,2,15,11,8,3,10,6,12,5,9,0,7,
                0,15,7,4,14,2,13,1,10,6,12,11,9,5,3,8,
                4,1,14,8,13,6,2,11,15,12,9,7,3,10,5,0,
                15,12,8,2,4,9,1,7,5,11,3,14,10,0,6,13
        },
        {
                15,1,8,14,6,11,3,4,9,7,2,13,12,0,5,10,
                3,13,4,7,15,2,8,14,12,0,1,10,6,9,11,5,
                0,14,7,11,10,4,13,1,5,8,12,6,9,3,2,15,
                13,8,10,1,3,15,4,2,11,6,7,12,0,5,14,9
        },


        {
                10,0,9,14,6,3,15,5,1,13,12,7,11,4,2,8,
                13,7,0,9,3,4,6,10,2,8,5,14,12,11,15,1,
                13,6,4,9,8,15,3,0,11,1,2,12,5,10,14,7,
                1,10,13,0,6,9,8,7,4,15,14,3,11,5,2,12
        },
        {
                7,13,14,3,0,6,9,10,1,2,8,5,11,12,4,15,
                13,8,11,5,6,15,0,3,4,7,2,12,1,10,14,9,
                10,6,9,0,12,11,7,13,15,1,3,14,5,2,8,4,
                3,15,0,6,10,1,13,8,9,4,5,11,12,7,2,14
        },
        {
                2,12,4,1,7,10,11,6,8,5,3,15,13,0,14,9,
                14,11,2,12,4,7,13,1,5,0,15,10,3,9,8,6,
                4,2,1,11,10,13,7,8,15,9,12,5,6,3,0,14,
                11,8,12,7,1,14,2,13,6,15,0,9,10,4,5,3
        },
        {
                12,1,10,15,9,2,6,8,0,13,3,4,14,7,5,11,
                10,15,4,2,7,12,9,5,6,1,13,14,0,11,3,8,
                9,14,15,5,2,8,12,3,7,0,4,10,1,13,11,6,
                4,3,2,12,9,5,15,10,11,14,1,7,6,0,8,13
        },
        {
                4,11,2,14,15,0,8,13,3,12,9,7,5,10,6,1,
                13,0,11,7,4,9,1,10,14,3,5,12,2,15,8,6,
                1,4,11,13,12,3,7,14,10,15,6,8,0,5,9,2,
                6,11,13,8,1,4,10,7,9,5,0,15,14,2,3,12
        },
        {
                13,2,8,4,6,15,11,1,10,9,3,14,5,0,12,7,
                1,15,13,8,10,3,7,4,12,5,6,11,0,14,9,2,
                7,11,4,1,9,12,14,2,0,6,10,13,15,3,5,8,
                2,1,14,7,4,10,8,13,15,12,9,0,3,5,6,11
        }
};

__device__ int e_cuda[48] = {
        32, 1, 2, 3, 4, 5,
        4, 5, 6, 7, 8, 9,
        8, 9,10,11,12,13,
        12,13,14,15,16,17,
        16,17,18,19,20,21,
        20,21,22,23,24,25,
        24,25,26,27,28,29,
        28,29,30,31,32, 1,
};

__device__ unsigned int n_symbols =10;

__device__ unsigned char* hex_to_bin_c(const unsigned char p[], int l, unsigned char ap[], unsigned char *st) //hexadecimal to binary
{
    unsigned int Idx = blockIdx.x;
    int inc = 0;
    for(int i=0;i<l;i++)
    {
        int count = 0;
        if(p[Idx*16+i]>=48 && p[Idx*16+i]<=57)
        {
            int te=int(p[Idx*16+i])-48;
            while(te>0)
            {
                st[Idx*4+count]=(unsigned char)(te%2+48);
                te/=2;
                count++;
            }
            while(count!=4)
            {
                st[Idx*4+count] = '0';
                count++;
            }
            for(int j=3;j>=0;j--)
            {
                ap[Idx*64+inc]=st[Idx*4+j];
                inc ++;
            }
        }
        else
        {
            int te=p[i]-'A'+10;
            while(te>0)
            {
                st[Idx*4+count]=(unsigned char)(te%2+48);
                te/=2;
                count++;
            }
            for(int j=3;j>=0;j--)
            {
                ap[Idx*64+inc] = st[Idx*4+j];
                inc++;
            }
        }
    }
    return ap;
}

__device__ unsigned char *string_to_hex_c(const unsigned char input[], int len, unsigned char output[])
{
    unsigned int Idx = blockIdx.x;
    static const char* const lut = "0123456789ABCDEF";
    int count = 0;
    for (int i=0; i<len; ++i) {
        const unsigned char c = input[Idx*8+i];
        output[Idx*16+count] = (lut[c >> 4]);
        count++;
        output[Idx*16+count] = (lut[c & 15]);
        count++;
    }
    return output;
}


__device__ unsigned char* encrypt_cuda(unsigned char *text, int *key, unsigned char *text_en, int len, unsigned char *l,
                                       unsigned char *r, unsigned char *rtem, unsigned char * ep, unsigned char *xorout,
                                       unsigned char *sout, unsigned char *soutt, unsigned char *pc, unsigned char *text_perm)
{
    unsigned int Idx = blockIdx.x;
    int i,m, j,row,col,temp,round=16, count2=0, count3=0, count4=0, count5=0;


    for(m=0;m<64;m++){
        text_perm[Idx*64+m] = text[Idx*64+ip_cuda[m]-1];
    }

    for(m=0; m<len/2; m++)
    {
        l[Idx*32+m] = text_perm[Idx*64+m];
        r[Idx*32+m] = text_perm[Idx*64+m+(len/2)];
    }

    while(round--) {
        for (m = 0; m < len / 2; m++) {
            rtem[Idx*32+m] = r[Idx*32+m];
        }
        count2 = 0;
        count4 = 0;

        //the expansion P box
        for(i = 0; i < 48; i++){
            ep[Idx*48+i] = r[Idx*32+e_cuda[i]-1];
        }
        //Key xor with output of expansion p box
        for (i = 0; i < 48; i++) {
            xorout[Idx*48+count2] = (unsigned char) (((int(ep[Idx*64+i]) - 48) ^ key[(48*(16-round-1)+i)] + 48));
            count2++;
        }
        //sbox compression 48bit to 32 bit
        for (i = 0; i < 48; i += 6) {
            row = (int(xorout[Idx*48+i + 5]) - 48) + (int(xorout[Idx*48+i]) - 48) * 2;
            col = (int(xorout[Idx*48+i + 1]) - 48) * 8 + (int(xorout[Idx*48+i + 2]) - 48) * 4 + (int(xorout[Idx*48+i + 3]) - 48) * 2 +
                  (int(xorout[Idx*48+i + 4]) - 48);
            temp = s_cuda[i / 6][row][col];
            count3 = 0;
            while (temp > 0) {
                soutt[Idx*4+count3] = (unsigned char) (temp % 2 + 48);
                count3++;
                temp /= 2;
            }
            while (count3 < 4) {
                soutt[Idx*4+count3] = 48;
                count3++;
            }
            for (j = count3 - 1; j >= 0; j--) {
                sout[Idx*48 + count4] = soutt[Idx*4+j];
                count4++;
            }
        }

        //straight pbox that is permutation of the sbox output
        for (i = 0; i < 32; i++)
        {
            pc[Idx*32+i] = sout[Idx*48 + per_cuda[i] - 1];
        }
        //r="";
        count5 = 0;
        for (i = 0; i < 32; i++)
        {
            r[Idx*32+count5] = (unsigned char) (((int(pc[Idx*32+i]) - 48) ^ (int(l[Idx*32+i]) - 48)) + 48);
            count5++;
        }
        for (m = 0; m < (len/2); m++)
        {
            l[Idx*(len/2)+m] = rtem[Idx*(len/2)+m];
        }

    }

    for(m=0; m<len/2; m++)
    {
        text_perm[Idx*(len)+m]=l[Idx*(len/2)+m];
        text_perm[Idx*(len)+m+len/2]=r[Idx*(len/2)+m];
    }

    for(m=0; m<64; m++){
        text_en[Idx*64+m] = text_perm[Idx*64+fp_cuda[m]-1];
    }

    return text_en;
}


__device__ void generateCombination(unsigned int *v, unsigned int k, int i)
{
    unsigned int Idx = blockIdx.x;
    unsigned int n = n_symbols;
    unsigned int div;
    for (int j=0; j<(k-1);j++){
        div = n;
        for (int m=2; m<=(k-1)-j; m++ ){
            div = div * n;
        }
        v[Idx*(k)+j] = i / div;
        i = i % div;
    }
    v[Idx*(k)+k-1] = i % n;
    return;
}

__global__ void brute_force_cuda(unsigned char encrypted_text_c[], unsigned int len, int* key, int threads,
        unsigned char *attack_psw, unsigned char *attack_h, unsigned char *attack_b, unsigned int* test,
        unsigned char *first_part, unsigned char *second_part, unsigned char *swap_part,
        unsigned char * ep, unsigned char *xorout, unsigned char *sout, unsigned char *soutt, unsigned char *pc,
                                 unsigned char *st, unsigned char *text_perm)
{
    unsigned int Idx = blockIdx.x;

    bool compare = false;
    unsigned int l = len;
    unsigned int n = n_symbols;

    unsigned int size = n;
    for (int k = 2; k <= l; k++) {
        size = size * n;
    }

    unsigned int size_per_thread = size / threads;
    unsigned int start = Idx*size_per_thread;
    unsigned int finish = start + size_per_thread;

    if(Idx >= threads){return;}
    if(Idx == threads)
    {
        finish = size;
    }

    for(int i=start; i<finish; i++)
    {
        generateCombination(test, l, i);

        //Convert test to char  array
        for (int j =0; j<l; j++)
        {
            attack_psw[Idx*l+j] = (unsigned char)test[Idx*l+j] + '0';
        }


        attack_h = string_to_hex_c(attack_psw, l, attack_h);
        attack_b = hex_to_bin_c(attack_h, l*2, attack_b, st);
        attack_b = encrypt_cuda(attack_b, key, attack_b, l*8, first_part, second_part, swap_part, ep, xorout, sout, soutt, pc, text_perm);

        for(int m=0; m < l*8; m++)
        {
            if(attack_b[Idx*(l*8)+m] != encrypted_text_c[m])
            {
                break;
            }
            if( m == (l*8)-1)
            {
                compare = true;
            }
        }
        if (compare)
        {
            printf("From thread %d \n", blockIdx.x);
            for(int m=0; m < 64; m++){
                printf("%d", attack_b[Idx*64+m] -'0');
            }
            printf("\n");
            asm("trap;");
            compare = false;
        }
    }
    printf("Any match found");
    return;
}


int main(){
    // Initialization parameters
    int threads = 0;
	unsigned int len = 8;
    string text, text_hex, text_bin, key_hex, key_bin, n_threads;
    locale loc;

    pre:;
    std::cout << "Enter the 8 characters password (only number):\n";
    cin >> text;
    if(text.length()!=8) {
        cout<<"Enter all the characters\n";
        goto pre;
    }

    for(int i = 0; i < text.length(); i++) {
        if(!isdigit(text[i], loc)) {
            cout << "Password must contain only numbers!\n";
            goto pre;
        }
    }

    text_hex = string_to_hex(text);
    for(int i=0;i<16;i++)
    {
        if((text_hex[i]>='0'&& text_hex[i]<='9')||(text_hex[i]>='A'&& text_hex[i]<='F'))
            ;
        else
        {
            cout<<"Not a valid hexadecimal string\n";
            goto pre;
        }
    }

    pre3:;
    cout << "Set number of threads (0: set one thread):\n";
    cin >> n_threads;
    for(int i = 0; i < n_threads.length(); i++) {
        if(!isdigit(n_threads[i], loc)) {
            cout << "Number of threads must be integer!\n";
            goto pre3;
        }
    }
    if(atoi(n_threads.c_str()) != 0)
        threads = atoi(n_threads.c_str());

    if(threads == 0)
    	threads = 1;

    key_hex = "AABB09182736CCDD";
    if(key_hex.length()!=16)
    {
        cout<<"Enter all the key bits (in hexadecimal)\n";
        return 0;
    }
    for(int i=0;i<16;i++)
    {
        if((key_hex[i]>='0'&& key_hex[i]<='9')||(key_hex[i]>='A'&& key_hex[i]<='F'))
            ;
        else
        {
            cout<<"Not a valid hexadecimal key string\n";
            return 0;
        }
    }
    // Generation of the key
    text_bin = hex_to_bin(text_hex);
    key_bin = hex_to_bin(key_hex);

    int **key;
    key = keygen(key_bin);

    // from 2D to 1D
    unsigned  int height = 16;
    unsigned int width = 48;
    int *key_1D = new int [height*width];
    for (int h = 0; h < height; h++){
        for (int w = 0; w < width; w++)
            key_1D[width * h + w] = key[h][w];
    }

    cout << "Crypting...\n" << endl;

    unsigned char text_bin_c[text_bin.length()];
    for(int i=0; i < text_bin.length(); i++)
    {
        text_bin_c[i] = (unsigned char)text_bin[i];
    }

    unsigned int l = len * sizeof(unsigned char);
    unsigned char * encrypted_b_c = (unsigned char*)malloc(l*8);
    unsigned char *encrypted_binary_text_c = encrypt_c(text_bin_c, key, encrypted_b_c,64);
    cout << "Encrypted text: " << encrypted_binary_text_c << endl;

    // Start time
    struct timeval start, end;
    gettimeofday(&start, NULL);


    // Necessary to brute force cuda function
    unsigned char * compare_text;
    unsigned char * attack_psw;
    unsigned char * attack_h;
    unsigned char * attack_b;
    unsigned int * test;

    //Necessary to hex to bin function
    unsigned char * st;

    // Necessary to encrypt cuda function
    unsigned char * first_part;
    unsigned char * second_part;
    unsigned char * swap_part;
    unsigned char * ep;
    unsigned char * xorout;
    unsigned char * sout;
    unsigned char * soutt;
    unsigned char * pc;
    unsigned char * text_perm;

    hipMalloc(&compare_text, l*8);
    hipMemcpy(compare_text, encrypted_binary_text_c, l*8, hipMemcpyHostToDevice);
    hipMalloc(&attack_psw, l*threads);
    hipMalloc(&attack_h, l*2*threads);
    hipMalloc(&attack_b, l*8*threads);
    hipMalloc(&test, l*threads);

    hipMalloc(&st, l*threads/2);

    hipMalloc(&first_part, l*4*threads);
    hipMalloc(&second_part, l*4*threads);
    hipMalloc(&swap_part, l*4*threads);
    hipMalloc(&ep, l*8*threads);
    hipMalloc(&xorout, l*6*threads);
    hipMalloc(&sout, l*6*threads);
    hipMalloc(&soutt, l*threads/2);
    hipMalloc(&pc, l*4*threads);
    hipMalloc(&text_perm, l*8*threads);

    unsigned int size_key= width*height* sizeof(int);
    int * p_key_1D;
    hipMalloc(&p_key_1D, size_key);
    hipMemcpy(p_key_1D, key_1D, size_key, hipMemcpyHostToDevice);

    cout<<"Searching text...\n"<<endl;

    // call to kernel
    brute_force_cuda<<<threads,1>>>(compare_text, len, p_key_1D, threads, attack_psw, attack_h, attack_b, test,
            first_part, second_part, swap_part, ep, xorout, sout, soutt, pc, st, text_perm);

    hipDeviceSynchronize();

    // End time
    gettimeofday(&end, NULL);
    double time_tot = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;
    cout<<"Total time: "<< time_tot <<endl;

    // Free GPU space
    hipFree(compare_text);
    hipFree(attack_psw);
    hipFree(attack_h);
    hipFree(attack_b);
    hipFree(test);

    hipFree(first_part);
    hipFree(second_part);
    hipFree(swap_part);
    hipFree(ep);
    hipFree(xorout);
    hipFree(sout);
    hipFree(soutt);
    hipFree(pc);
    hipFree(text_perm);

    return 0;
}
